#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include<time.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <conio.h>
#define NUM_NODES 6
#define num_blks 1
#define half_NUMNODES 4
#define Thread 5
int CPU_NODE = half_NUMNODES;
bool front[NUM_NODES] = { false };
bool visited[NUM_NODES] = { false };


typedef struct
{
	int start;     // Index of first adjacent neigbour node in d_adjLists	
	int length;    // Number of neighbour nodes 
} Node;


__global__ void CUDA_BFS_KERNEL(Node* d_VertixArray, int* d_adjLists, bool* d_front, bool* d_Visited, bool* done, int* d_result)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > NUM_NODES)
		*done = false;


	if (d_front[id] == true && d_Visited[id] == false)
	{
		printf("%d ", id); //This printf gives the order of vertices in BFS	
		d_front[id] = false;
		d_Visited[id] = true;
		__syncthreads();
		//	int k = 0;
			//int i;
		int start = d_VertixArray[id].start;
		int end = start + d_VertixArray[id].length;
		for (int i = start; i < end; i++)
		{
			int nid = d_adjLists[i];

			if (d_Visited[nid] == false && d_front[nid] == false)
			{
				//printf("%d", nid);
				d_front[nid] = true;

				*done = false;
			}

		}

	}
}


void CPU_BFS(Node* Vertix, int* adjLists, bool* front, bool* Visited, bool done)
{
	done = false;

	for (int i = half_NUMNODES; i < NUM_NODES;i++) {

		if (front[i] == true && Visited[i] == false)
		{

			printf("%d", i);
			front[i] = false;
			Visited[i] = true;


			int start = Vertix[i].start;
			int end = start + Vertix[i].length;
			for (int j = start; j < end; j++)
			{
				int nid = adjLists[j];

				if (Visited[nid] == false && front[nid] == false)
				{
					printf(" %d", nid);
					front[nid] = true;
					done = false;

				}

			}

		}

	}
}


int main()
{
	Node Vertex[NUM_NODES];
	int edges[14];
	int GPU_edges[7];
	int* adjLists = (int*)malloc(sizeof(int*) * 7);
	int* result[half_NUMNODES];
	hipEvent_t start, stop;
	Node* d_VertexArray;
	int* d_adjLists;
	int* d_result;
	bool done;
	bool* d_done;
	bool* d_front;
	bool* d_Visited;



	Vertex[0].start = 0;
	Vertex[0].length = 2;

	Vertex[1].start = 2;
	Vertex[1].length = 2;

	Vertex[2].start = 4;
	Vertex[2].length = 3;

	Vertex[3].start = 7;
	Vertex[3].length = 3;

	Vertex[4].start = 10;
	Vertex[4].length = 2;

	Vertex[5].start = 12;
	Vertex[5].length = 2;


	GPU_edges[0] = 1;
	GPU_edges[1] = 2;
	GPU_edges[2] = 0;
	GPU_edges[3] = 3;
	GPU_edges[4] = 0;
	GPU_edges[5] = 3;
	GPU_edges[6] = 5;
	edges[7] = 1;
	edges[8] = 2;
	edges[9] = 4;
	edges[10] = 3;
	edges[11] = 5;
	edges[12] = 2;
	edges[13] = 4;




	int source = 0;
	front[source] = true;


	hipMalloc((void**)&d_VertexArray, sizeof(Node) * half_NUMNODES);
	hipMemcpy(d_VertexArray, Vertex, sizeof(Node) * half_NUMNODES, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_adjLists, sizeof(Node) * half_NUMNODES);
	hipMemcpy(d_adjLists, GPU_edges, sizeof(Node) * half_NUMNODES, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_front, sizeof(bool) * half_NUMNODES);
	hipMemcpy(d_front, front, sizeof(bool) * half_NUMNODES, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_Visited, sizeof(bool) * half_NUMNODES);
	hipMemcpy(d_Visited, visited, sizeof(bool) * half_NUMNODES, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_done, sizeof(bool));
	hipMalloc((void**)&d_result, sizeof(int*) * NUM_NODES);
	hipMemcpy(d_result, result, sizeof(int) * half_NUMNODES, hipMemcpyHostToDevice);
	int count = 0;
	printf("\nBreadth-First Search: ");
	//printf("%d", source);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	do {
		count++;
		done = true;
		hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		CUDA_BFS_KERNEL << <num_blks, Thread >> > (d_VertexArray, d_adjLists, d_front, d_Visited, d_done, d_result);
		hipMemcpy(&done, d_done, sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(&visited, d_Visited, sizeof(bool) * half_NUMNODES, hipMemcpyDeviceToHost);
		hipMemcpy(&result, d_result, sizeof(int) * half_NUMNODES, hipMemcpyDeviceToHost);

	} while (!done);


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	int x = 0;
	for (int i = 0; i < 7; i++) {
		x = GPU_edges[i];
		front[x] = true;
		//printf("\ni =%d , x =%d , visited = %d\n", i, x, visited[x]);
	}


	for (int c = 0; c < NUM_NODES; c++) {
		printf("\ni =%d , d_visited =%d , visited = %d, front =%d, d_front =%d \n  in bfs \n", c, &d_Visited[c], visited[c], front[c], &d_front[c]);
	}





	//int CPU_SourceNode = half_NUMNODES;
	//front[CPU_SourceNode] = true;


	for (int i = 7; i < 14;i++) {
		adjLists[i] = edges[i];

	}


	do {

		done = true;
		CPU_BFS(Vertex, adjLists, front, visited, done);

	} while (!done);

	printf("\nGPU Time: %f s \n", elapsedTime / 1000);
	hipFree(d_done);
	hipFree(d_VertexArray);
	hipFree(d_adjLists);
	hipFree(d_front);
	hipFree(d_Visited);
	hipFree(d_result);

}